
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <fstream>
#include "time.h"


using namespace std;

__host__ void preprocesamientoKMP(char* pattern, int m, int f[])
{
    int k;
    f[0] = -1;
    for (int i = 1; i < m; i++){
        k = f[i - 1];
        while (k >= 0){
            if (pattern[k] == pattern[i - 1])
                break;
            else
                k = f[k];
        }
        f[i] = k + 1;
    }
}
 
//check whether target string contains pattern 
__global__ void KMP(char* pattern, char* target,int f[],int c[],int sizePattern, int sizeText)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    int i = sizePattern * index;
    int j = sizePattern * (index + 2)-1;

    //printf("1-i: %i j: %i n: %i index: %i\n", i, j, sizePattern, index);

    if(i > sizeText)
        return;
    if(j > sizeText)
        j = sizeText;

    //printf("2-i: %i j: %i n: %i index: %i\n", i, j, sizePattern, index);

    int k = 0;        
    while (i < j)
    {
        if (k == -1)
        {
            i++;
            k = 0;
        }
        else if (target[i] == pattern[k])
        {
            i++;
            k++;
            if (k == sizePattern)
            {
                c[i - sizePattern] = i - sizePattern;
                i = i - k + 1;
            }
        }
        else
            k = f[k];
    }
    return;
}
 
int main(int argc, char* argv[])
{
    // constante de tamaño
    const int S = 40000000;
    
    // cantidad de threads
    int M = 1024;

    // controla tamaño de char 1 a 4
    int charSize = 4;

    // varibles en CPU
    char *tar;
    char *pat;
    tar = (char*)malloc(2000000);
    pat = (char*)malloc(S*charSize);
    
    // Variables en GPU
    char *d_tar;
    char *d_pat;

    // Stream Files
    ifstream inputFileText;
    ifstream inputFilePattern;
    ofstream outputFileText;

    // Abrir archivos
    inputFileText.open(argv[1]);
    inputFilePattern.open(argv[2]);
    outputFileText.open("DATA/result.txt");

    inputFileText>>tar;
    inputFilePattern>>pat;

    int m = strlen(tar);
    int n = strlen(pat);
    int *fault;
    int *coin;

    fault = new int[m];
    coin = new int[m];

    int *d_fault;
    int *d_coin;

    // inicializar arreglo c con -1 para procesamiento y resultados
    for(int i = 0;i<m; i++)
        coin[i] = -1;


    //num blocks
    int blocks = (m/n+M)/M;

    printf("Copiando datos a GPU\n");
    
    //time_t timeL_init, timeL_end, timeT_end, timeT_init;
    hipEvent_t start, stop, local_s, local_e;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&local_s);
    hipEventCreate(&local_e);
    
    preprocesamientoKMP(pat, m, fault);
    hipEventRecord(start);
    /* Crear variables en cuda */
    hipMalloc((void **)&d_tar, m*charSize);
    hipMalloc((void **)&d_pat, n*charSize);
    hipMalloc((void **)&d_fault, m*charSize);
    hipMalloc((void **)&d_coin, m*charSize);

    /* Copia de datos a GPU */
    hipMemcpy(d_tar, tar, m*charSize, hipMemcpyHostToDevice);
    hipMemcpy(d_pat, pat, n*charSize, hipMemcpyHostToDevice);
    hipMemcpy(d_fault, fault, m*charSize, hipMemcpyHostToDevice);
    hipMemcpy(d_coin, coin, m*charSize, hipMemcpyHostToDevice);
    
    hipEventRecord(local_s);
    KMP<<<blocks,M>>>(d_pat, d_tar ,d_fault, d_coin, n, m);
    hipEventRecord(local_e);

    hipMemcpy(coin, d_coin, m*charSize, hipMemcpyDeviceToHost);

    // liberar memoria de GPU
    hipFree(d_tar);
    hipFree(d_pat);
    hipFree(d_fault);
    hipFree(d_coin);
    hipEventRecord(stop);

    float milis, local;
    hipEventSynchronize(stop);
    hipEventSynchronize(local_e);
    hipEventElapsedTime(&milis, start, stop);
    hipEventElapsedTime(&local, local_s, local_e);
    
    // mostrar resultados
    for(int i = 0;i<m; i++)
        if(coin[i]!=-1)
            outputFileText<<"position: "<<i<<"\tmatch: "<<coin[i]<<'\n';
    
    
    printf("Blocks: %i Threads: %i n: %i m:%i\n", (m/n+M)/M, M, n, m);
    printf("Tiempo ejecucion: %1.15f ml.\n", milis);
    printf("Tiempo ejecucion kernel: %1.15f ml.\n", local);

    return 0;
}
